#include <iostream>
#include <mpi.h>
#include <hip/hip_runtime.h>

int main(int argc, char *argv[]) {
    MPI_Init(&argc, &argv);
    MPI_Comm world = MPI_COMM_WORLD;

    int n_devices;
    hipGetDeviceCount(&n_devices);

    int rank;
    MPI_Comm_rank(world, &rank);
    int dev_id = rank % n_devices;

    std::cout << "Rank " << rank << " is using device "
    << dev_id << std::endl;

    MPI_Finalize();
    return 0;
}